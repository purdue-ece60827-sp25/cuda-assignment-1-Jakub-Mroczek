#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	int globalID = blockIdx.x*blockDim.x + threadIdx.x;

	if(globalID < size) {
		y[globalID] = x[globalID]*scale + y[globalID];
	}
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	float scale = 2.0f;

	int blockSize = 256;
	int gridSize = (vectorSize + blockSize -1)/blockSize;

	float *h_x = (float *) malloc(vectorSize * sizeof(float));
	float *h_y = (float *) malloc(vectorSize * sizeof(float));
	float *h_saxpy_result = (float *) malloc(vectorSize * sizeof(float));

	if (h_x == NULL || h_y == NULL || h_saxpy_result == NULL) {
		printf("Unable to malloc host memory ... Exiting!");
		return -1;
	}

	vectorInit(h_x, vectorSize);
	vectorInit(h_y, vectorSize);

	float *d_x = nullptr;
	float *d_y = nullptr;

	hipMalloc(&d_x, vectorSize*sizeof(float));
	hipMalloc(&d_y, vectorSize*sizeof(float));

	if (!d_x || !d_y) {
		printf("Unable to malloc device memory ... Exiting!");
		return -1;
	}

	hipMemcpy(d_x, h_x, vectorSize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, vectorSize*sizeof(float), hipMemcpyHostToDevice);

	#ifndef DEBUG_PRINT_DISABLE 
		printf("\n Adding vectors : \n");
		printf(" scale = %f\n", scale);
		printf(" x = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", h_x[i]);
		}
		printf(" ... }\n");
		printf(" y = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", h_y[i]);
		}
		printf(" ... }\n");
	#endif

	saxpy_gpu<<<gridSize, blockSize>>>(d_x, d_y, scale, vectorSize);
	hipMemcpy(h_saxpy_result, d_y, vectorSize*sizeof(float), hipMemcpyDeviceToHost);

	#ifndef DEBUG_PRINT_DISABLE 
		printf(" saxpy_result = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", h_saxpy_result[i]);
		}
		printf(" ... }\n");
	#endif

	int errorCount = verifyVector(h_x, h_y, h_saxpy_result, scale, vectorSize);
	std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";

	std::cout << "runGpuSaxpy complete\n";

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	int globalID = blockIdx.x*blockDim.x + threadIdx.x;

	hiprandState_t rng;
	hiprand_init(clock64(), globalID, 0, &rng);

	if(globalID < pSumSize){
		uint64_t hit_count = 0;
		for(uint64_t i = 0; i<sampleSize; i++){
			float x = hiprand_uniform(&rng);
			float y = hiprand_uniform(&rng);
			if(int(x * x + y * y) == 0){
				hit_count++;
			}
		}
		pSums[globalID] = hit_count;
	}
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	int globalID = blockIdx.x*blockDim.x + threadIdx.x;

	if(globalID*reduceSize < pSumSize){
		int start = globalID*reduceSize;
		int end = min(start+reduceSize, pSumSize);
		int hit_total = 0;
		for(int i=start; i<end; i++){
			hit_total = hit_total + pSums[i];
		}
		totals[globalID] = hit_total;
	}

}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	int blockSize = 256;
	int gridSize = (generateThreadCount + blockSize -1)/blockSize;

	double approxPi = 0;

	uint64_t totalsLength = (generateThreadCount + reduceSize -1)/reduceSize;
	// uint64_t * h_pSums = (uint64_t *) malloc(generateThreadCount*sizeof(uint64_t));
	uint64_t * h_totals = (uint64_t *) malloc(totalsLength*sizeof(uint64_t));
	uint64_t * d_pSums = nullptr;
	uint64_t * d_totals = nullptr;
	hipMalloc(&d_pSums, generateThreadCount*sizeof(uint64_t));
	hipMalloc(&d_totals, totalsLength*sizeof(uint64_t));
	
	generatePoints<<<gridSize, blockSize>>>(d_pSums, generateThreadCount, sampleSize);
	reduceCounts<<<gridSize, blockSize>>>(d_pSums, d_totals, generateThreadCount, reduceSize);
	hipMemcpy(h_totals, d_totals, totalsLength*sizeof(uint64_t), hipMemcpyDeviceToHost);

	uint64_t hitcount = 0;
	for(int i = 0; i<totalsLength; i++){
		hitcount = hitcount + h_totals[i];
	}

	#ifndef DEBUG_PRINT_DISABLE 
		std::cout << "hitcount = " << hitcount << "\n";
	#endif

	approxPi = (double)hitcount/(generateThreadCount*sampleSize);
	approxPi = approxPi*4.0;

	return approxPi;
}
